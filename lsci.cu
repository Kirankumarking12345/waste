#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <time.h>

int timer_flag = 0;
struct timeval start_time, end_time;
double elapsed_time;

#define wi 1920
#define h 1200
#define BUFFER_SIZE wi *h

typedef struct
{
  uint8_t r;
  uint8_t g;
  uint8_t b;
} rgb;

/********************LUT Colorspce*************************/
uint8_t lookup_table[256][3] =
    {
        {48, 18, 59},
        {49, 21, 66},
        {50, 24, 74},
        {52, 27, 81},
        {53, 30, 88},
        {54, 33, 95},
        {55, 35, 101},
        {56, 38, 108},
        {57, 41, 114},
        {58, 44, 121},
        {59, 47, 127},
        {60, 50, 133},
        {60, 53, 139},
        {61, 55, 145},
        {62, 58, 150},
        {63, 61, 156},
        {64, 64, 161},
        {64, 67, 166},
        {65, 69, 171},
        {65, 72, 176},
        {66, 75, 181},
        {67, 78, 186},
        {67, 80, 190},
        {67, 83, 194},
        {68, 86, 199},
        {68, 88, 203},
        {69, 91, 206},
        {69, 94, 210},
        {69, 96, 214},
        {69, 99, 217},
        {70, 102, 221},
        {70, 104, 224},
        {70, 107, 227},
        {70, 109, 230},
        {70, 112, 232},
        {70, 115, 235},
        {70, 117, 237},
        {70, 120, 240},
        {70, 122, 242},
        {70, 125, 244},
        {70, 127, 246},
        {70, 130, 248},
        {69, 132, 249},
        {69, 135, 251},
        {69, 137, 252},
        {68, 140, 253},
        {67, 142, 253},
        {66, 145, 254},
        {65, 147, 254},
        {64, 150, 254},
        {63, 152, 254},
        {62, 155, 254},
        {60, 157, 253},
        {59, 160, 252},
        {57, 162, 252},
        {56, 165, 251},
        {54, 168, 249},
        {52, 170, 248},
        {51, 172, 246},
        {49, 175, 245},
        {47, 177, 243},
        {45, 180, 241},
        {43, 182, 239},
        {42, 185, 237},
        {40, 187, 235},
        {38, 189, 233},
        {37, 192, 230},
        {35, 194, 228},
        {33, 196, 225},
        {32, 198, 223},
        {30, 201, 220},
        {29, 203, 218},
        {28, 205, 215},
        {27, 207, 212},
        {26, 209, 210},
        {25, 211, 207},
        {24, 213, 204},
        {24, 215, 202},
        {23, 217, 199},
        {23, 218, 196},
        {23, 220, 194},
        {23, 222, 191},
        {24, 224, 189},
        {24, 225, 186},
        {25, 227, 184},
        {26, 228, 182},
        {27, 229, 180},
        {29, 231, 177},
        {30, 232, 175},
        {32, 233, 172},
        {34, 235, 169},
        {36, 236, 166},
        {39, 237, 163},
        {41, 238, 160},
        {44, 239, 157},
        {47, 240, 154},
        {50, 241, 151},
        {53, 243, 148},
        {56, 244, 145},
        {59, 244, 141},
        {63, 245, 138},
        {66, 246, 135},
        {70, 247, 131},
        {74, 248, 128},
        {77, 249, 124},
        {81, 249, 121},
        {85, 250, 118},
        {89, 251, 114},
        {93, 251, 111},
        {97, 252, 108},
        {101, 252, 104},
        {105, 253, 101},
        {109, 253, 98},
        {113, 253, 95},
        {116, 254, 92},
        {120, 254, 89},
        {124, 254, 86},
        {128, 254, 83},
        {132, 254, 80},
        {135, 254, 77},
        {139, 254, 75},
        {142, 254, 72},
        {146, 254, 70},
        {149, 254, 68},
        {152, 254, 66},
        {155, 253, 64},
        {158, 253, 62},
        {161, 252, 61},
        {164, 252, 59},
        {166, 251, 58},
        {169, 251, 57},
        {172, 250, 55},
        {174, 249, 55},
        {177, 248, 54},
        {179, 248, 53},
        {182, 247, 53},
        {185, 245, 52},
        {187, 244, 52},
        {190, 243, 52},
        {192, 242, 51},
        {195, 241, 51},
        {197, 239, 51},
        {200, 238, 51},
        {202, 237, 51},
        {205, 235, 52},
        {207, 234, 52},
        {209, 232, 52},
        {212, 231, 53},
        {214, 229, 53},
        {216, 227, 53},
        {218, 226, 54},
        {221, 224, 54},
        {223, 222, 54},
        {225, 220, 55},
        {227, 218, 55},
        {229, 216, 56},
        {231, 215, 56},
        {232, 213, 56},
        {234, 211, 57},
        {236, 209, 57},
        {237, 207, 57},
        {239, 205, 57},
        {240, 203, 58},
        {242, 200, 58},
        {243, 198, 58},
        {244, 196, 58},
        {246, 194, 58},
        {247, 192, 57},
        {248, 190, 57},
        {249, 188, 57},
        {249, 186, 56},
        {250, 183, 55},
        {251, 181, 55},
        {251, 179, 54},
        {252, 176, 53},
        {252, 174, 52},
        {253, 171, 51},
        {253, 169, 50},
        {253, 166, 49},
        {253, 163, 48},
        {254, 161, 47},
        {254, 158, 46},
        {254, 155, 45},
        {254, 152, 44},
        {253, 149, 43},
        {253, 146, 41},
        {253, 143, 40},
        {253, 140, 39},
        {252, 137, 38},
        {252, 134, 36},
        {251, 131, 35},
        {251, 128, 34},
        {250, 125, 32},
        {250, 122, 31},
        {249, 119, 30},
        {248, 116, 28},
        {247, 113, 27},
        {247, 110, 26},
        {246, 107, 24},
        {245, 104, 23},
        {244, 101, 22},
        {243, 99, 21},
        {242, 96, 20},
        {241, 93, 19},
        {239, 90, 17},
        {238, 88, 16},
        {237, 85, 15},
        {236, 82, 14},
        {234, 80, 13},
        {233, 77, 13},
        {232, 75, 12},
        {230, 73, 11},
        {229, 70, 10},
        {227, 68, 10},
        {226, 66, 9},
        {224, 64, 8},
        {222, 62, 8},
        {221, 60, 7},
        {219, 58, 7},
        {217, 56, 6},
        {215, 54, 6},
        {214, 52, 5},
        {212, 50, 5},
        {210, 48, 5},
        {208, 47, 4},
        {206, 45, 4},
        {203, 43, 3},
        {201, 41, 3},
        {199, 40, 3},
        {197, 38, 2},
        {195, 36, 2},
        {192, 35, 2},
        {190, 33, 2},
        {187, 31, 1},
        {185, 30, 1},
        {182, 28, 1},
        {180, 27, 1},
        {177, 25, 1},
        {174, 24, 1},
        {172, 22, 1},
        {169, 21, 1},
        {166, 20, 1},
        {163, 18, 1},
        {160, 17, 1},
        {157, 16, 1},
        {154, 14, 1},
        {151, 13, 1},
        {148, 12, 1},
        {145, 11, 1},
        {142, 10, 1},
        {139, 9, 1},
        {135, 8, 1},
        {132, 7, 1},
        {129, 6, 2},
        {125, 5, 2},
        {122, 4, 2}};
/*********************************************************/
/*

__global__ void fillZeroPaddedArray(uint16_t *buffer, float *zimg, int w)
{
  // int half_w = floor(w / 2);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("%d vale i\n",i);
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < h + 4 && j < wi + 4)
  {
    if ((i > 1 && j > 1) && (i < h + 2 && j < wi + 2))
    {
      zimg[i * (wi + 4) + j] = (float)buffer[(i - 2) * wi + (j - 2)];
    }
    else
    {
      zimg[i * (wi + 4) + j] = (float)255;
    }
  }
}
*/
__global__ void lsci_kernel(uint16_t *buffer, float *zimg, float *Z, float *maxVal)
{
  float s = 0;
  float m = 0.0;
  float sd = 0.0;
  float SD = 0.0;
  float x = 0;
  int w = 5;
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("%d vale i\n",i);
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < h + 4 && j < wi + 4)
  {
    if ((i > 1 && j > 1) && (i < h + 2 && j < wi + 2))
    {
      zimg[i * (wi + 4) + j] = (float)buffer[(i - 2) * wi + (j - 2)];
    }
    else
    {
      zimg[i * (wi + 4) + j] = (float)255;
    }
  }
  
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i >= 2 && i < h + 2) && (j >= 2 && j < wi + 2))
  {
    // printf("hello");
    s = 0;

    s = zimg[(i - 2) * (wi + 4) + (j - 2)] + zimg[(i - 2) * (wi + 4) + (j - 1)] + zimg[(i - 2) * (wi + 4) + j] + zimg[(i - 2) * (wi + 4) + (j + 1)] + zimg[(i - 2) * (wi + 4) + (j + 2)] +
        zimg[(i - 1) * (wi + 4) + (j - 2)] + zimg[(i - 1) * (wi + 4) + (j - 1)] + zimg[(i - 1) * (wi + 4) + j] + zimg[(i - 1) * (wi + 4) + (j + 1)] + zimg[(i - 1) * (wi + 4) + (j + 2)] +
        zimg[i * (wi + 4) + (j - 2)] + zimg[i * (wi + 4) + (j - 1)] + zimg[i * (wi + 4) + j] + zimg[i * (wi + 4) + (j + 1)] + zimg[i * (wi + 4) + (j + 2)] +
        zimg[(i + 1) * (wi + 4) + (j - 2)] + zimg[(i + 1) * (wi + 4) + (j - 1)] + zimg[(i + 1) * (wi + 4) + j] + zimg[(i + 1) * (wi + 4) + (j + 1)] + zimg[(i + 1) * (wi + 4) + (j + 2)] +
        zimg[(i + 2) * (wi + 4) + (j - 2)] + zimg[(i + 2) * (wi + 4) + (j - 1)] + zimg[(i + 2) * (wi + 4) + j] + zimg[(i + 2) * (wi + 4) + (j + 1)] + zimg[(i + 2) * (wi + 4) + (j + 2)];

    m = s / 25;
    SD = 0;

    SD =  (zimg[(i - 2) * (wi + 4) + (j - 2)] - m) * (zimg[(i - 2) * (wi + 4) + (j - 2)] - m)
        + (zimg[(i - 2) * (wi + 4) + (j - 1)] - m) * (zimg[(i - 2) * (wi + 4) + (j - 1)] - m)
        + (zimg[(i - 2) * (wi + 4) + j] - m) * (zimg[(i - 2) * (wi + 4) + j] - m)
        + (zimg[(i - 2) * (wi + 4) + (j + 1)] - m) * (zimg[(i - 2) * (wi + 4) + (j + 1)] - m)
        + (zimg[(i - 2) * (wi + 4) + (j + 2)] - m) * (zimg[(i - 2) * (wi + 4) + (j + 2)] - m)

        + (zimg[(i - 1) * (wi + 4) + (j - 2)] - m) * (zimg[(i - 1) * (wi + 4) + (j - 2)] - m)
        + (zimg[(i - 1) * (wi + 4) + (j - 1)] - m) * (zimg[(i - 1) * (wi + 4) + (j - 1)] - m)
        + (zimg[(i - 1) * (wi + 4) + j] - m) * (zimg[(i - 1) * (wi + 4) + j] - m)
        + (zimg[(i - 1) * (wi + 4) + (j + 1)] - m) * (zimg[(i - 1) * (wi + 4) + (j + 1)] - m)
        + (zimg[(i - 1) * (wi + 4) + (j + 2)] - m) * (zimg[(i - 1) * (wi + 4) + (j + 2)] - m)

        + (zimg[i * (wi + 4) + (j - 2)] - m) * (zimg[i * (wi + 4) + (j - 2)] - m)
        + (zimg[i * (wi + 4) + (j - 1)] - m) * (zimg[i * (wi + 4) + (j - 1)] - m)
        + (zimg[i * (wi + 4) + j] - m) * (zimg[i * (wi + 4) + j] - m)
        + (zimg[i * (wi + 4) + (j + 1)] - m) * (zimg[i * (wi + 4) + (j + 1)] - m)
        + (zimg[i * (wi + 4) + (j + 2)] - m) * (zimg[i * (wi + 4) + (j + 2)] - m)

        + (zimg[(i + 1) * (wi + 4) + (j - 2)] - m) * (zimg[(i + 1) * (wi + 4) + (j - 2)] - m)
        + (zimg[(i + 1) * (wi + 4) + (j - 1)] - m) * (zimg[(i + 1) * (wi + 4) + (j - 1)] - m)
        + (zimg[(i + 1) * (wi + 4) + j] - m) * (zimg[(i + 1) * (wi + 4) + j] - m)
        + (zimg[(i + 1) * (wi + 4) + (j + 1)] - m) * (zimg[(i + 1) * (wi + 4) + (j + 1)] - m)
        + (zimg[(i + 1) * (wi + 4) + (j + 2)] - m) * (zimg[(i + 1) * (wi + 4) + (j + 2)] - m)

        + (zimg[(i + 2) * (wi + 4) + (j - 2)] - m) * (zimg[(i + 2) * (wi + 4) + (j - 2)] - m)
        + (zimg[(i + 2) * (wi + 4) + (j - 1)] - m) * (zimg[(i + 2) * (wi + 4) + (j - 1)] - m)
        + (zimg[(i + 2) * (wi + 4) + j] - m) * (zimg[(i + 2) * (wi + 4) + j] - m)
        + (zimg[(i + 2) * (wi + 4) + (j + 1)] - m) * (zimg[(i + 2) * (wi + 4) + (j + 1)] - m)
        + (zimg[(i + 2) * (wi + 4) + (j + 2)] - m) *  (zimg[(i + 2) * (wi + 4) + (j + 2)] - m) ;

        
     /*   
    for (int k = i - 2; k < i + w - 2; k++)
    {
      for (int l = j - 2; l < j + w - 2; l++)
      {
        x = (zimg[k * (wi + 4) + l] > m) ? zimg[k * (wi + 4) + l] - m : m - zimg[k * (wi + 4) + l];
        SD = SD + x * x;
      }
    }
    */
    sd = SD / (w * w);

    sd = sqrt(sd);

    Z[(i - 2) * (wi) + (j - 2)] = sd / m;

    // atomicMax(maxVal, Z[(i - 2) * (wi) + (j - 2)]);

    // printf("%f ", *maxVal);

    /*
    if (Z[(i - 2) * (wi ) + (j - 2)] > *maxVal)
    {
        *maxVal = Z[(i - 2) * (wi) + (j - 2)];
    }
    printf("%f ", *maxVal);
    */
  }
}

__global__ void normalize_invert(float *Z, float max)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < h && j < wi)
  {
    Z[i * wi + j] = 1 - (Z[i * wi + j] / max); // Normalize & invert the image
  }
}


__global__ void grayscale_to_rgb_kernel(float *Z, uint8_t *lookup_table, rgb *buffer_rgb) {
    float range_min = 0.86;
  float range_max = 0.93;
  float range = range_max - range_min;
  float rangeinv = 1 / range;
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < 1200 && idy < 1920) {
        float grey = (Z[idx * 1920 + idy] > range_min) ? ((Z[idx * 1920 + idy] < range_max) ? ((Z[idx * 1920 + idy] - range_min) * rangeinv * 255) : 255) : 0;
        buffer_rgb[idx * 1920 + idy].b = lookup_table[(int)grey * 3 + 2];
        buffer_rgb[idx * 1920 + idy].g = lookup_table[(int)grey * 3 + 1];
        buffer_rgb[idx * 1920 + idy].r = lookup_table[(int)grey * 3];
    }
}

int img = 0;
char name[50];
int main(int argc, char **argv)
{
  gettimeofday(&start_time, NULL);
  uint16_t buffer[BUFFER_SIZE];

  uint16_t *a;
  float *zimg;
  float *Z;

  // buffer = (uint16_t *)malloc(BUFFER_SIZE* sizeof(uint16_t));
  a = (uint16_t *)malloc(h * wi * sizeof(uint16_t));
  Z = (float *)malloc((wi) * (h) * sizeof(float));
  zimg = (float *)malloc((wi + 4) * (h + 4) * sizeof(float));

  for (img = 1; img <= 1; img++)
  {
    FILE *ptr;
    sprintf(name, "/home/ayati/Desktop/lsci/data/RemoveBG_%d.bin", img);
    // printf("%s\n", name);
    ptr = fopen(name, "rb");
    if (!ptr)
    {
      printf("Unable to open file!");
      // return 1;
    }
    fread(buffer, sizeof(buffer), 1, ptr);
    fclose(ptr);

    int w = 5;
    int half_w = floor(w / 2);

    uint16_t *dev_buffer;
    float *dev_zimg;
    
    hipMalloc((void **)&dev_buffer, sizeof(uint16_t) * h * wi);
    hipMemcpy(dev_buffer, buffer, sizeof(uint16_t) * h * wi, hipMemcpyHostToDevice);
    /*cudaMalloc((void **)&dev_zimg, sizeof(float) * (h + 4) * (wi + 4));

    // Copy input array from host to device
    cudaMemcpy(dev_buffer, buffer, sizeof(uint16_t) * h * wi, cudaMemcpyHostToDevice);

    // Define number of threads and blocks for the kernel
    dim3 threadsPerBlock(5, 5);
    dim3 numBlocks((h + 4 + threadsPerBlock.x - 1) / threadsPerBlock.x, (wi + 4 + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Call kernel function
    fillZeroPaddedArray<<<numBlocks, threadsPerBlock>>>(dev_buffer, dev_zimg, w);

    // Wait for kernel to finish
    cudaDeviceSynchronize();

    // Copy zero-padded array from device to host
    cudaMemcpy(zimg, dev_zimg, sizeof(float) * (h + 4) * (wi + 4), cudaMemcpyDeviceToHost);

    // Free memory on the GPU
    cudaFree(dev_buffer);
    cudaFree(dev_zimg);
*/
    /*
           FILE *f;
      f = fopen("kiran.pgm", "wb");
      if (!f) {
        printf("Unable to create file.\n");
        return;
      }

      fprintf(f, "P5\n%d %d\n255\n", wi+4, h+4);

      for (int i = 0; i < h+4; i++) // h-2
      {
        for (int j = 0; j < wi+4; j++)
        {
           // Normalise & invert the image
          fputc(zimg[i * (wi+4) + j],f);
          //fputc(a[i * wi + j], f);
        }
      }
      fclose(f);
      */

    float *d_zimg, *d_Z;
    float *d_maxVal;
    float maxVal = 0;
    hipMalloc((void **)&d_zimg, sizeof(float) * (h + 4) * (wi + 4));
    hipMalloc((void **)&d_Z, sizeof(float) * (h + 4) * (wi + 4));

    hipMemcpy(d_maxVal, &maxVal, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_zimg, zimg, sizeof(float) * (h + 4) * (wi + 4), hipMemcpyHostToDevice);

    dim3 threadsBlock(5, 5);
    dim3 num_Blocks((h + threadsBlock.x - 1) / threadsBlock.x, (wi + threadsBlock.y - 1) / threadsBlock.y);

    lsci_kernel<<<num_Blocks, threadsBlock>>>(dev_buffer,d_zimg, d_Z, d_maxVal);

    hipMemcpy(Z, d_Z, sizeof(float) * (h + 4) * (wi + 4), hipMemcpyDeviceToHost);
    hipMemcpy(&maxVal, d_maxVal, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_zimg);
    hipFree(d_Z);

    /*finding max*/
    float max = 0;
    for (int i = 0; i < h; i++) // h-2
    {
      for (int j = 0; j < wi; j++)
      {
        // if(Z[(i) * (wi) + (j)] > 1)printf("%f ", Z[(i) * (wi) + (j)]);
        if (Z[(i) * (wi) + (j)] > max)
          max = Z[(i) * (wi) + (j)]; 
      }
    }

     //printf("%f ", max);
//max = 1.5;
    hipMalloc(&d_Z, h * wi * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_Z, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with appropriate grid and block size
    dim3 blockDim(5, 5);
    dim3 gridDim((h + blockDim.x - 1) / blockDim.x, (wi + blockDim.y - 1) / blockDim.y);
    normalize_invert<<<gridDim, blockDim>>>(d_Z, max);

    // Copy output data from device to host
    hipMemcpy(Z, d_Z, h * wi * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(d_Z);

uint8_t *d_lookup_table;
rgb *d_buffer_rgb;

rgb *buffer_rgb = (rgb *)malloc(1920 * 1200 * sizeof(rgb));
  hipMalloc(&d_Z, 1200 * 1920 * sizeof(float));
  hipMalloc(&d_lookup_table, 256 * 3 * sizeof(uint8_t));
  hipMalloc(&d_buffer_rgb, 1200 * 1920 * sizeof(rgb));

  hipMemcpy(d_Z, Z, 1200 * 1920 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_lookup_table, lookup_table, 256 * 3 * sizeof(uint8_t), hipMemcpyHostToDevice);

  dim3 block_size(32, 32);
  dim3 grid_size((1200 + block_size.x - 1) / block_size.x, (1920 + block_size.y - 1) / block_size.y);
  grayscale_to_rgb_kernel<<<grid_size, block_size>>>(d_Z, d_lookup_table, d_buffer_rgb);

  hipMemcpy(buffer_rgb, d_buffer_rgb, 1200 * 1920 * sizeof(rgb), hipMemcpyDeviceToHost);

  // Use buffer_rgb here

  hipFree(d_Z);
  hipFree(d_lookup_table);
  hipFree(d_buffer_rgb);

  FILE *fp = fopen("color_output.ppm", "wb");
  fprintf(fp, "P3\n");
  fprintf(fp, "1920 1200\n");
  fprintf(fp, "255\n");
  for (int i = 0; i < 1920 * 1200; i++) {
    fprintf(fp, "%d %d %d ", buffer_rgb[i].r, buffer_rgb[i].g, buffer_rgb[i].b);
  }
  fclose(fp);

   }

    gettimeofday(&end_time, NULL);
          elapsed_time = (end_time.tv_sec - start_time.tv_sec) +
                         (end_time.tv_usec - start_time.tv_usec) / 1000000.0;

          printf("The time elapsed is :%f", elapsed_time);
    FILE *f;
    f = fopen("kiran.pgm", "wb");
    if (!f)
    {
      printf("Unable to create file.\n");
      return;
    }

    fprintf(f, "P5\n%d %d\n255\n", wi, h);

    for (int i = 0; i < h; i++) // h-2
    {
      for (int j = 0; j < wi; j++)
      {
        // Normalise & invert the image
        fputc(Z[i * (wi) + j] * 255, f);
        // fputc(a[i * wi + j], f);
      }
    }
    fclose(f);

   
   #ifdef solve 
  cudaDeviceReset();
#endif
  return 0;
}